#include "hip/hip_runtime.h"
#include "emt6ro/cell/cell.h"
#include <cmath>
#include <hip/hip_runtime_api.h>

namespace emt6ro {

namespace {

__host__ __device__ Cell::CyclePhase progressPhase(Cell::CyclePhase current, bool change) {
  return static_cast<Cell::CyclePhase>(static_cast<uint8_t>(current) +
                                       static_cast<uint8_t>(change));
}

}  // namespace

__host__ __device__ void Cell::metabolise(Substrates &site_substrates,
                                          const Parameters::Metabolism &metabolism) {
  site_substrates -= metabolism.values[static_cast<uint8_t>(mode)];
}

__host__ __device__ bool Cell::progressClock(float time_step) {
  if ((mode == MetabolicMode::AEROBIC_PROLIFERATION ||
      mode == MetabolicMode::ANAEROBIC_PROLIFERATION) &&
      time_in_repair == 0) {
    proliferation_time += time_step / 3600;
    CyclePhase current = phase;
    phase = progressPhase(current,
                          proliferation_time > cycle_times.times[static_cast<uint8_t>(current)]);
    return phase != current && phase != CyclePhase::D;
  }
  return false;
}
__host__ __device__ bool Cell::tryProliferating(const Substrates &levels,
                                                const Parameters &params) {
  if (levels.cho >= params.metabolism.aerobic_proliferation.cho &&
      levels.ox >= params.metabolism.aerobic_proliferation.ox &&
      levels.gi < params.quiescence_gi) {
    mode = MetabolicMode::AEROBIC_PROLIFERATION;
    return true;
  } else if (levels.cho >= params.metabolism.anaerobic_proliferation.cho &&
             levels.ox >= params.metabolism.anaerobic_proliferation.ox &&
             levels.gi < params.quiescence_gi) {
    mode = MetabolicMode::ANAEROBIC_PROLIFERATION;
    return true;
  }
  return false;
}

__host__ __device__ bool Cell::tryQuiescence(const Substrates &levels, const Parameters &params) {
  if (phase != CyclePhase::G1 && phase != CyclePhase::G2) {
    return false;
  }
  if (levels.cho >= params.metabolism.aerobic_quiescence.cho &&
      levels.ox >= params.metabolism.aerobic_quiescence.ox &&
      levels.gi < params.death_gi) {
    mode = MetabolicMode::AEROBIC_QUIESCENCE;
    return true;
  } else if (levels.cho >= params.metabolism.anaerobic_quiescence.cho &&
      levels.ox >= params.metabolism.anaerobic_quiescence.ox &&
      levels.gi < params.death_gi) {
    mode = MetabolicMode::ANAEROBIC_QUIESCENCE;
    return true;
  }
  return false;
}

__host__ __device__ bool Cell::enterG1SStopping(float time_step, uint8_t vacant_neighbours) {
  return proliferation_time > (cycle_times.g1 - 2*time_step / 3600.f) &&
         phase == CyclePhase::G1 && vacant_neighbours <= 1;
}

__host__ __device__ bool Cell::updateState(const Substrates &levels, const Parameters &params,
                                           uint8_t vacant_neighbors, uint8_t &meta) {
  if (proliferation_time >= cycle_times.d - params.time_step / 3600.f)
    return false;
  if (!enterG1SStopping(params.time_step, vacant_neighbors)) {
    if (tryProliferating(levels, params)) {
      return true;
    }
  }
  bool r = tryQuiescence(levels, params);
  if (!r) {
    meta = levels.ox < params.metabolism.aerobic_quiescence.ox;
    meta |= (uint8_t)(levels.cho < params.metabolism.anaerobic_quiescence.cho) << 1U;
    meta |= (uint8_t)mode << 2U;
  }
  return r;
}

__host__ __device__ void Cell::irradiate(float dose, const Parameters::CellRepair &params) {
  irradiation = irradiation / (1 + time_in_repair / params.repair_half_time) + dose;
  calcDelayTime(params);
}

void Cell::calcDelayTime(const Parameters::CellRepair &params) {
  using std::exp;
  repair_delay_time = params.delay_time.coeff * exp(params.delay_time.exp_coeff * irradiation);
}

}  // namespace emt6ro
