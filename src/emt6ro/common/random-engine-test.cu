#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "emt6ro/common/random-engine.h"

namespace emt6ro {

__global__ void fillUniform(float *data, hiprandState_t *states) {
  auto x = blockIdx.x * blockDim.x + threadIdx.x;
  CuRandEngine engine(&states[x]);
  data[x] = engine.uniform();
}

TEST(RandomGenerator, FillUniform) {
  device::buffer<float> data(2 * 1024);
  std::vector<uint32_t> h_seeds(2*1024);
  for (size_t i = 0; i < 2 * 1024; ++i) h_seeds[i] = i;
  auto d_seeds = device::buffer<uint32_t>::fromHost(h_seeds.data(), 2*1024);
  CuRandEngineState state(2 * 1024, d_seeds.data());
  fillUniform<<<2, 1024>>>(data.data(), state.states());
  auto h_data = data.toHost();
  for (size_t i = 0; i < 2 * 1024; ++i) {
    ASSERT_TRUE(h_data[i] >= 0.0 && h_data[i] <= 1.0);
  }
}

}  // namespace emt6ro