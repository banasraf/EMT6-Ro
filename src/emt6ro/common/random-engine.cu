#include "hip/hip_runtime.h"
#include "emt6ro/common/random-engine.h"
#include "emt6ro/common/cuda-utils.h"

namespace emt6ro {

namespace detail {

__global__ void initializeState(hiprandState_t *state, uint32_t seed, size_t size) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) return;
  hiprand_init(seed, i, 0, &state[i]);
}

void init(hiprandState_t *state_data, uint32_t seed, size_t size, hipStream_t stream) {
  auto mbs = CuBlockDimX * CuBlockDimY;
  auto blocks = div_ceil(size, mbs);
  auto block_size = (size > mbs) ? mbs : size;
  detail::initializeState<<<blocks, block_size, 0, stream>>>(state_data, seed, size);
}

}  // namespace detail

CuRandEngineState::CuRandEngineState(size_t size, uint32_t seed, hipStream_t stream) : state_(size) {
  init(seed, stream);
}

CuRandEngineState::CuRandEngineState(size_t size): state_(size) {}

void CuRandEngineState::init(uint32_t seed, hipStream_t stream) {
  detail::init(state_.data(), seed, state_.size(), stream);
}

__device__ float CuRandEngine::uniform() {
  return hiprand_uniform(state);
}

__device__ float CuRandEngine::normal(const Parameters::NormalDistribution& params) {
  return  params.stddev * hiprand_normal(state) + params.mean;
}

}  // namespace emt6ro
