#include "hip/hip_runtime.h"
#include "emt6ro/common/random-engine.h"

namespace emt6ro {

namespace detail {

__global__ void initializeState(hiprandState_t *state, const uint32_t *seeds, size_t size) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) return;
  hiprand_init(seeds[i], 0, 0, &state[i]);
}

void init(hiprandState_t *state_data, const uint32_t *seeds, size_t size, hipStream_t stream) {
  size_t block_size = (size > 1024) ? 1024 : size;
  size_t grid_size = (size + block_size - 1) / block_size;
  detail::initializeState<<<grid_size, block_size, 0, stream>>>(state_data, seeds, size);
}

}  // namespace detail

CuRandEngineState::CuRandEngineState(size_t size, const uint32_t* seeds) : state_(size) {
  init(seeds);
}

CuRandEngineState::CuRandEngineState(size_t size): state_(size) {}

void CuRandEngineState::init(const uint32_t *seeds, hipStream_t stream) {
  detail::init(state_.data(), seeds, state_.size(), stream);
}

__device__ float CuRandEngine::uniform() {
  return hiprand_uniform(state);
}

__device__ float CuRandEngine::normal(const Parameters::NormalDistribution& params) {
  return  params.stddev * hiprand_normal(state) + params.mean;
}

}  // namespace emt6ro
