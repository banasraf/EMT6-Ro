#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <iostream>
#include <random>
#include <vector>
#include <cmath>
#include "emt6ro/common/debug.h"
#include "emt6ro/common/grid.h"
#include "emt6ro/diffusion/diffusion.h"
#include "emt6ro/simulation/cell-division.h"
#include "emt6ro/simulation/simulation.h"
#include "emt6ro/statistics/statistics.h"
#include "emt6ro/common/cuda-utils.h"
#include "emt6ro/common/stack.cuh"
#include "emt6ro/common/error.h"

namespace emt6ro {

namespace detail {

__global__ void populateGridViewsKernel(GridView<Site> *views, uint32_t batch_size,
                                        Dims dims, Site *origin) {
  auto idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < batch_size) {
    views[idx].data = origin + dims.vol() * idx;
    views[idx].dims = dims;
  }
}

__host__ __device__ uint8_t vacantNeighbours(const GridView<Site> &grid, int16_t r, int16_t c) {
  return grid(r - 1, c - 1).isVacant() +
         grid(r - 1, c + 1).isVacant() +
         grid(r + 1, c - 1).isVacant() +
         grid(r + 1, c + 1).isVacant() +
         grid(r, c - 1).isVacant() +
         grid(r, c + 1).isVacant() +
         grid(r - 1, c).isVacant() +
         grid(r + 1, c).isVacant();
}

static const int kFindOccupiedNthreads = 32;
__global__ void findOccupied(GridView<Site> *lattices, uint32_t *occupied_b) {
  extern __shared__ uint32_t shmem[];
  auto lattice = lattices[blockIdx.x];
  uint32_t n = 0;
  Coords collection[1024 / kFindOccupiedNthreads];
  GRID_FOR(0, 0, lattice.dims.height - 1, lattice.dims.width - 1) {
    if (lattice(r, c).isOccupied()) {
      collection[n++] = Coords{r, c};
    }
  }
  shmem[threadIdx.x] = n;
  __syncthreads();
  uint32_t acc = 0;
  for (int i = 0; i < threadIdx.x; ++i) {
    acc += shmem[i];
  }
  uint32_t &n_occupied = occupied_b[blockIdx.x * 1024];
  auto *occupied = reinterpret_cast<Coords*>(&n_occupied + 1);
  for (int i = 0; i < n; ++i)
    occupied[acc + i] = collection[i];
  if (threadIdx.x == blockDim.x - 1)
    n_occupied = acc + n;
}

__global__ void cellSimulationKernel(GridView<Site> *grids, uint32_t *occupied_b,
                                     Parameters params, hiprandState_t *rand_states,
                                     Protocol *protocols, uint32_t step) {
  extern __shared__ uint64_t shm[];
  StackView<Coords> occupied(&occupied_b[blockIdx.x * 1024]);
  uint64_t division = 0;
  uint8_t vacant_neighbours[4];
  auto &grid = grids[blockIdx.x];
  const auto &protocol = protocols[blockIdx.x];
  hiprandState_t *rand_state =
      rand_states + blockDim.x * blockIdx.x + threadIdx.x;
  CuRandEngine rand(rand_state);
  uint8_t subi = 0;
  for (auto coords : dev_iter(occupied)) {
    vacant_neighbours[subi] = vacantNeighbours(grid, coords.r, coords.c);
    ++subi;
  }
  __syncthreads();
  subi = 0;
  auto dose = protocol.getDose(step);
  for (auto coords : dev_iter(occupied)) {
    auto &site = grid(coords);
    auto d = site.step(params, vacant_neighbours[subi], dose, rand);
    if (d) {
      auto child_coords = chooseNeighbour(coords.r, coords.c, rand);
      if (grid(child_coords).isVacant()) {
        division = Coords2(coords, child_coords).encode();
      }
    }
    ++subi;
  }
  division = block_reduce(division, shm,
                          [](uint64_t a, uint64_t b){return a | (b * (uint64_t)(!a));});
  if (threadIdx.x == 0 && division) {
    Coords2 coords = Coords2::decode(division);
    auto parent = coords[0];
    auto child = coords[1];
    grid(child).state = Site::State::OCCUPIED;
    grid(child).cell = divideCell(grid(parent).cell, params, rand);
    occupied.push(child);
  }
}

}  // namespace detail

void Simulation::populateLattices() {
  auto mbs = CuBlockDimX * CuBlockDimY;
  auto blocks = div_ceil(batch_size, mbs);
  auto block_size = (batch_size > mbs) ? mbs : batch_size;
  detail::populateGridViewsKernel<<<blocks, block_size, 0, str.stream_>>>
    (lattices.data(), batch_size, dims, data.data());
  KERNEL_DEBUG("populate")
}

Simulation::Simulation(uint32_t batch_size, const Parameters &parameters, uint32_t seed)
    : batch_size(batch_size)
    , dims(Dims(parameters.lattice_dims.height+2, parameters.lattice_dims.width+2))
    , params(parameters)
    , data(batch_size * dims.vol())
    , protocols(batch_size)
    , lattices(batch_size)
    , rois(batch_size)
    , border_masks(batch_size * dims.vol())
    , occupied(batch_size * 1024)
    , rand_state(batch_size * simulate_num_threads)
    , results(batch_size) {
  rand_state.init(seed, str.stream_);
  populateLattices();
}

void Simulation::sendData(const HostGrid<Site> &grid, const Protocol &protocol, uint32_t multi) {
  assert(filled_samples + multi <= batch_size);
  assert(grid.view().dims == dims);
  for (uint32_t i = filled_samples; i < filled_samples + multi; ++i) {
    auto view = grid.view();
    data.copyHost(view.data, dims.vol(), dims.vol() * i, str.stream_);
    KERNEL_DEBUG("data")
    protocols.copyHost(&protocol, 1, i, str.stream_);
    KERNEL_DEBUG("protocol")
  }
  filled_samples += multi;
  filled_protocols += multi;
}

void Simulation::step() {
  if (step_ % 128 == 0) {
    detail::findOccupied
    <<<batch_size, detail::kFindOccupiedNthreads,
       detail::kFindOccupiedNthreads * sizeof(uint32_t), str.stream_>>>
    (lattices.data(), occupied.data());
  }
  if (step_ % 32 == 0) {
    updateROIs();
  }
  diffuse();
  simulateCells();
  ++step_;
}

void Simulation::diffuse() {
  batchDiffusion(lattices.data(), rois.data(), border_masks.data(), params.diffusion_params,
                 params.external_levels, params.time_step/params.diffusion_params.time_step,
                 dims, batch_size, str.stream_);
}

void Simulation::simulateCells() {
  detail::cellSimulationKernel
    <<<batch_size, simulate_num_threads, sizeof(uint64_t)*simulate_num_threads/32, str.stream_>>>
    (lattices.data(), occupied.data(), params, rand_state.states(),
     protocols.data(), step_);
  KERNEL_DEBUG("simulate cells")
}

void Simulation::updateROIs() {
  findROIs(rois.data(), border_masks.data(), lattices.data(), batch_size, str.stream_);
}

void Simulation::getResults(uint32_t *h_results) {
  countLiving(results.data(), data.data(), dims, batch_size, str.stream_);
  hipMemcpyAsync(h_results, results.data(), batch_size * sizeof(uint32_t),
                  hipMemcpyDeviceToHost, str.stream_);
  sync();
}

void Simulation::run(uint32_t nsteps) {
  ENFORCE(filled_samples == batch_size, "");
  for (uint32_t s = 0; s < nsteps; ++s) {
    step();
  }
}

void Simulation::getData(Site *h_data, uint32_t sample) {
  ENFORCE(sample < batch_size, make_string("Cannot read data from sample ",
                                           sample, ". Batch size: ", batch_size));
  hipMemcpyAsync(h_data, data.data() + sample * dims.vol(),
            dims.vol() * sizeof(Site), hipMemcpyDeviceToHost, str.stream_);
  sync();
}

void Simulation::sync() {
  hipStreamSynchronize(str.stream_);
}

void Simulation::reset() {
  sync();
  step_ = 0;
  filled_samples = 0;
}

void Simulation::setState(const Site *state) {
  hipMemcpyAsync(data.data(), state, batch_size * dims.vol() * sizeof(Site), 
                  hipMemcpyDeviceToDevice, str.stream_);
  KERNEL_DEBUG("copy data");
  filled_samples = batch_size;
}

void Simulation::setProtocols(const Protocol *ps) {
  hipMemcpyAsync(protocols.data(), ps, batch_size * sizeof(Protocol), 
                  hipMemcpyHostToDevice, str.stream_);
  KERNEL_DEBUG("copy protocols");
  filled_protocols = batch_size;
}

}  // namespace emt6ro
